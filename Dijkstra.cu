#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <time.h>
#include <cstdio>

#define UNKNOWN_DISTANCE -1
#define UNKNOWN_NODE -2
#define NO_EDGE_BEWTEEN -3
#define GENEDGE_HAS_PATH_TO_START 0
#define GENEDGE_NO_PATH_TO_START -1
#define MAXIMUM_WEIGHT 100
#define NODE_VISITED 1
#define NODE_UNVISITED 0
#define INT32_MAX 2147483647

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    //std::cout << "gpuErrchk: " << hipGetErrorString(code) << std::endl;
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int DijkstraSelectNode(int nodeNum, int* visited, int* disFromStart){
    int curClosestNode = -1;
    int curClosestDistance = INT32_MAX;
    for(int i=0; i<nodeNum; i++){
        if((visited[i] == NODE_UNVISITED) 
           && (disFromStart[i] != UNKNOWN_DISTANCE)
           && (disFromStart[i] < curClosestDistance)) {
            curClosestNode = i;
            curClosestDistance = disFromStart[i];
        }
    }
    return curClosestNode;
}
__global__ void divide100(int nodeNum,int *visited_addr,int *dis_addr,int *max_addr,int *index_addr)
{
    int b = nodeNum/100;
    int start = threadIdx.x * b;

    int curClosestNode = -1;
    int curClosestDistance = INT32_MAX;
    for(int i=start; i<start+b; i++){
        if((visited_addr[i] == NODE_UNVISITED)  && (dis_addr[i] != UNKNOWN_DISTANCE)  && (dis_addr[i] < curClosestDistance)) {
            curClosestNode = i;
            curClosestDistance = dis_addr[i];
        }
    }
    //printf("thread %d closest dis = %d\n",threadIdx.x,curClosestDistance);
    max_addr[threadIdx.x]=curClosestDistance;
    index_addr[threadIdx.x]=curClosestNode;
}

__global__ void select(int nodeNum,int *visited_addr,int *max_addr,int *index_addr,int *cur_addr)
{
    int cur=0;
    int curClosestDistance = INT32_MAX;
    for(int i=0;i<100;i++)
    {
        //printf("i=%d max=%d\n",i,max_addr[i]);
        if(max_addr[i]<curClosestDistance)
        {
            curClosestDistance = max_addr[i];
            cur=index_addr[i];
        }
    }
    *cur_addr = cur;
    visited_addr[cur]=NODE_VISITED;
    //printf("current=%d\n",cur);
}

__global__ void selectall(int nodeNum,int *visited_addr,int *dis_addr,int *cur_addr)
{
    int cur=0;
    int curClosestDistance = INT32_MAX;
    for(int i=0;i<nodeNum;i++)
    {
        if((visited_addr[i] == NODE_UNVISITED)  && (dis_addr[i] != UNKNOWN_DISTANCE)  && (dis_addr[i] < curClosestDistance)) 
        {
            curClosestDistance = dis_addr[i];
            cur=i;
        }
    }
    *cur_addr = cur;
    visited_addr[cur]=NODE_VISITED;
    //printf("current=%d\n",cur);
}
__global__ void test( int *disFromStart,int nodeNum)
{
    for(int i =0 ; i<nodeNum;i++)
    {
        printf("node %d dis = %d",i,disFromStart[i]);
    }
}
__global__ void init(int* visited_addr, int *dis_addr, int* prevNode_addr,int nodeNum )
{
    int i = blockIdx.x*1024 + threadIdx.x;
    if(i<nodeNum) //boundery check
    {    
        if(i==0)
        {
            dis_addr[i]=0;
            prevNode_addr[i]=0;
        }
        else
        {
            visited_addr[i]=NODE_UNVISITED;
            dis_addr[i]=UNKNOWN_DISTANCE;
            prevNode_addr[i]=-2;
        }
    }
}

__global__ void visit_update(int* visited_addr, int *cur_addr)
{
    visited_addr[*cur_addr]=NODE_VISITED;
} 
__global__ void update( int *edges, int* visited, int *disFromStart,int *prevNode, int* cur_addr ,int nodeNum )
{
    int    i = blockIdx.x*blockDim.x + threadIdx.x;
    int curNode = *cur_addr;
    if(i<nodeNum) //boundery check
    {        
        if((visited[i] != NODE_VISITED) && (i != curNode) )
        {
            if(edges[i+curNode*nodeNum] != NO_EDGE_BEWTEEN)
            {
                if ( (disFromStart[i] == UNKNOWN_DISTANCE)  || (disFromStart[i] > (disFromStart[curNode] + edges[i+curNode*nodeNum])) )
                {
                    disFromStart[i] = (disFromStart[curNode] + edges[i+curNode*nodeNum]);
                    prevNode[i] = curNode;
                }
            }
        }
    } 
}
void DijkstraMain(int nodeNum, int edgeNum,void * e, int* disFromStart, int* prevNode){
    clock_t ini,inid,start,divi,mid,u;
    ini = clock();
    std::cout <<"start disj\n"  << std::endl;
    int (*edges)[nodeNum] = (int (*)[nodeNum])e;
    // Initialize visited list
    int block = (nodeNum-1)/1024 +1;
    int size = block*1024*sizeof(int);

    //int *visited = (int*)malloc(sizeof(int) *  block * 1024);
    int *edges_addr;
    int *dis_addr, *visited_addr, *prevNode_addr;//size
    int *max_addr,*index_addr;//100
    int *cur_addr;
    
    float selectt, updatet;
    selectt = 0.0;
    updatet = 0.0;
    gpuErrchk(hipMalloc(&dis_addr,size));
    gpuErrchk(hipMalloc(&visited_addr,size));
    gpuErrchk(hipMalloc(&prevNode_addr,size));
    gpuErrchk(hipMalloc(&cur_addr,sizeof(int))); 
    gpuErrchk(hipMalloc(&max_addr,100*sizeof(int)));
    gpuErrchk(hipMalloc(&index_addr,100*sizeof(int)));
    gpuErrchk(hipMalloc(&edges_addr,nodeNum* nodeNum * sizeof(int))); //nodenum * size of int x nodenum
    //inid = clock();
    //printf("malloc= %lf", ((double) (inid - ini)) / CLOCKS_PER_SEC);
    //hipMemset(edges_addr,0x01,pitch* nodeNum * sizeof(int));
    gpuErrchk(hipMemcpy(edges_addr,edges, nodeNum* nodeNum * sizeof(int), hipMemcpyHostToDevice));
    //ini = clock();
    //printf("memcpy= %lf", ((double) (ini - inid)) / CLOCKS_PER_SEC);
    //gpuErrchk(hipMemcpy(visited_addr,visited,size,hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(prevNode_addr,prevNode,size,hipMemcpyHostToDevice));
    //test<<<1,1024>>>();
    //hipDeviceSynchronize();
    init<<<block,1024>>>(visited_addr, dis_addr,prevNode_addr,nodeNum); 
    //hipDeviceSynchronize();
    inid = clock();
    //printf("init func= %lf", ((double) (inid - ini)) / CLOCKS_PER_SEC);
    //std::cout <<"init complete";
    for (int i=0; i<nodeNum; i++) {
        //int curNode = DijkstraSelectNode(nodeNum, visited, disFromStart); 
        //start = clock();
        divide100<<<1,100>>>(nodeNum,visited_addr,dis_addr,max_addr,index_addr);
        //divi = clock();
        select<<<1,1>>>(nodeNum,visited_addr,max_addr,index_addr,cur_addr);
        //selectall<<<1,1>>>(nodeNum,visited_addr,dis_addr,cur_addr);
        //mid = clock();
        //save to current node in cuda
        update<<<block,1024>>>(edges_addr, visited_addr, dis_addr, prevNode_addr, cur_addr, nodeNum);
        //u  = clock();
        //gpuErrchk(hipMemcpy(disFromStart,dis_addr,size,hipMemcpyDeviceToHost));
        //visit_update<<<1,1>>>(visited_addr,cur_addr);
        
        //printf("divide= %lf select= %lf update= %lf   \n", ((double) (divi - start)) / CLOCKS_PER_SEC,((double) (mid - divi)) / CLOCKS_PER_SEC,((double) (u - mid)) / CLOCKS_PER_SEC);
        //selectt += ((double) (mid - divi)) / CLOCKS_PER_SEC;
        //updatet += ((double) (u - mid)) / CLOCKS_PER_SEC;
        //printf("round %d   \n",i);
        //printf("dijk = %lf \n", ((double) (end - start)) / CLOCKS_PER_SEC);
    }
    ini = clock();
    printf("for looop= %lf", ((double) (ini - inid)) / CLOCKS_PER_SEC);
    //printf("select= %lf, update= %lf\n", selectt, updatet);
    gpuErrchk(hipMemcpy(disFromStart,dis_addr,size,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(prevNode,prevNode_addr,size,hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(dis_addr));
    gpuErrchk(hipFree(visited_addr));
    gpuErrchk(hipFree(prevNode_addr));
    gpuErrchk(hipFree(edges_addr));
    //free(visited);
    //inid = clock();
    //printf("free= %lf", ((double) (inid - ini)) / CLOCKS_PER_SEC);
}

int main(int argc, char* argv[]){
    srand(time(NULL));
    // Check the number of nodes and average number of edges are provided.
    if(argc < 2){
        printf("Usage: %s [num of nodes] [num of edges] (option: --display-graph)\n", argv[0]);
        return 0;
    }
    if(std::atoi(argv[2]) < (std::atoi(argv[1]) - 1)){
        printf("%s: (num of edges) must greater than (num of nodes - 1)\n", argv[0]);
        return 0;
    }

    int nodeNum = std::atoi(argv[1]);
    int edgeNum = std::atoi(argv[2]);

    // Read input
    std::fstream input;
    input.open("graph", std::fstream::in | std::fstream::binary);
    int *edges = (int*)malloc(sizeof(int*) * nodeNum * nodeNum);
    //int edges[nodeNum][nodeNum];
    for (int i=0; i<nodeNum; i++){
        for (int j=i; j<nodeNum; j++){
            int tmp;
            input.read((char *)&tmp, sizeof(int));
            edges[i*nodeNum+j] = tmp;
            edges[j*nodeNum+i] = tmp;
        }
    }
    // ************************** //
    // *** DIJKSTRA ALGORITHM *** //
    // ************************** //
    printf("efge complete ");
    int block = (nodeNum-1)/1024 +1;
    // Initialize node list.
    // The node list is the distance to each node from node 0(which is the beginning.
    int *disFromStart = (int*)malloc(sizeof(int) * block * 1024);
    for (int i=0; i<nodeNum; i++) disFromStart[i] = UNKNOWN_DISTANCE;
    disFromStart[0] = 0;

    // Initialize previous node list.
    // The previous node list record the last node of each shortest path to each node.
    int *prevNode = (int*)malloc(sizeof(int) * block * 1024);
    for (int i=0; i<nodeNum; i++) prevNode[i] = UNKNOWN_NODE;
    prevNode[0] = 0;


    clock_t start , end;
	start = clock();
    double cpu_time_used;
    
    DijkstraMain(nodeNum, edgeNum, (void *)edges, disFromStart, prevNode);

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("cuda = %lf \n", cpu_time_used);
    FILE* timeout;
    timeout = fopen("cuda_time.txt", "a");
    fprintf(timeout, "%lf\n", cpu_time_used);
    // ************************************ //
    // *** DIJKSTRA ALGORITHM COMPLETED *** //
    // ************************************ //


    // DEBUG: print the edge matrix
    //if(argc == 4){
    //    for (int i=0; i<nodeNum; i++) {
    //        for (int j=0; j<nodeNum; j++) {
    //            printf("%3d ",edges[i*nodeNum+j] == NO_EDGE_BEWTEEN ? 0 : edges[i*nodeNum+j]);
    //        }
    //        printf("\n");
    //    }
    //}

    // Output the shortest distance of every node from start.
    //printf("Shortest distance from start:\n");
    //for (int i=0; i<nodeNum; i++) {
     //   printf("%3d ",disFromStart[i]);
    //}

    // Output the last node of every node's shortest path.
    //printf("\n");
    //printf("\nPrevious node:\n");
    //for (int i=0; i<nodeNum; i++) {
    //    printf("%3d ",prevNode[i]);
    //}
    //printf("\n");


    // End of the program.
    free(disFromStart);
    free(prevNode);
    //for (int i=0; i<nodeNum; i++) free(edges[i]);
    //free(edges);
    return 0;
}